#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void block_and_thread() {
  printf("blockIdx (%d, %d, %d) threadIdx (%d, %d, %d)\n", 
          blockIdx.x, blockIdx.y, blockIdx.z,
          threadIdx.x, threadIdx.y, threadIdx.z);
}

int main() {
  // dim3 (x, y, z)
  dim3 grid(3, 2, 1);  // total 6 blocks (grid of blocks)
  dim3 block(3);       // 3 threads per block (18 threads in total)
  block_and_thread<<<grid, block>>>();

  // wait for above operation to complete
  hipDeviceSynchronize();

  // reset cuda devide
  hipDeviceReset();

  return 0;
}