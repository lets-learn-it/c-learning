#include "hip/hip_runtime.h"
#include <stdio.h>

int main() {
  int device_id = 0; // select the first CUDA device
  hipDeviceProp_t device_prop;
  hipGetDeviceProperties(&device_prop, device_id);

  // Query the maximum number of blocks in each dimension
  printf("Maximum blocks per grid:\n");
  printf("x dimension: %d\n", device_prop.maxGridSize[0]);
  printf("y dimension: %d\n", device_prop.maxGridSize[1]);
  printf("z dimension: %d\n", device_prop.maxGridSize[2]);

  // Query the maximum number of threads per block in each dimension
  printf("Maximum threads per block:\n");
  printf("x dimension: %d\n", device_prop.maxThreadsDim[0]);
  printf("y dimension: %d\n", device_prop.maxThreadsDim[1]);
  printf("z dimension: %d\n", device_prop.maxThreadsDim[2]);

  return 0;
}