/* #include "cuda.h" */  // this is low level api
#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void hello_world() {
  printf("Hello World");
}

int main() {
  // <<<1,1>>> are kernel lauch parameters
  // this function is asynchronous (will not wait)
  // <<<number_of_blocks, threads_per_block>>>
  hello_world<<<1,1>>>();

  // wait for above operation to complete
  hipDeviceSynchronize();

  // reset cuda devide / destroy CUDA context
  // All device allocations are removed.
  hipDeviceReset();

  return 0;
}